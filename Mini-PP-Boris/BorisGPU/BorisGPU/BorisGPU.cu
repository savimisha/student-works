#include "hip/hip_runtime.h"
#include "BorisGPU.cuh"
#include "Vector3.cuh"


__constant__ double LIGHTSPEED = 29979245800.0;
__constant__ double PI = 3.14159265358979323846;
__constant__ double EPS = 0.05;


__host__ __device__ double Det(Vector3 &a, Vector3 &b, Vector3 &c)
{
	return a.x*(b.y*c.z-b.z*c.y)-b.x*(a.y*c.z-a.z*c.y)+c.x*(a.y*b.z-a.z*b.y);
}

__global__ void Boris_kernel(Particle *parts, Vector3 E, Vector3 B, double dt)
{
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	Vector3 p_old, u_old, u_minus, u_new, p_new, V_new, r_new;
	Vector3 a,b,c,d;
	double Y_old, delta, factor;
	Vector3 u_plus;
	p_old = (parts[j].m/sqrt(1-(parts[j].V.SqNorm()/(LIGHTSPEED*LIGHTSPEED))))*parts[j].V;
	u_old = (1/(parts[j].m*LIGHTSPEED))*p_old;
	Y_old = sqrt(1+u_old.SqNorm());
	factor = parts[j].q*(dt)/(2*Y_old*LIGHTSPEED*parts[j].m);
	u_minus = u_old+(parts[j].q*dt/(2*parts[j].m*LIGHTSPEED))*E;
			
	a.x = 1;
	a.y = factor*B.z;
	a.z = -factor*B.y;
	b.x = -factor*B.z;
	b.y = 1;
	b.z = factor*B.x;
	c.x = factor*B.y;
	c.y = -factor*B.x;
	c.z = 1;
	d.x = u_minus.x+factor*(u_minus.y*B.z-u_minus.z*B.y);
	d.y = u_minus.y+factor*(u_minus.z*B.x-u_minus.x*B.z);
	d.z = u_minus.z+factor*(u_minus.x*B.y-u_minus.y*B.x);

	delta = Det(a,b,c);
	u_plus.x = Det(d,b,c)/delta;
	u_plus.y = Det(a,d,c)/delta;
	u_plus.z = Det(a,b,d)/delta;
	
	u_new = u_plus+factor*Y_old*E;
	p_new = parts[j].m*LIGHTSPEED*u_new;
	V_new = 1/(parts[j].m*sqrt(1+p_new.SqNorm()/(parts[j].m*parts[j].m*LIGHTSPEED*LIGHTSPEED)))*p_new;
	r_new = parts[j].r+V_new*dt;
	parts[j].r = r_new;
	parts[j].V = V_new;
		
}

void BorisGPU(Particle *parts, Vector3 E, Vector3 B, double dt, int N, int Count, int BlockSize)
{

	int nb = Count * sizeof(Particle);
	Particle *partsDev = NULL; 
	
	hipMalloc((void**)&partsDev, nb);
	
	hipMemcpy( partsDev, parts, nb, hipMemcpyHostToDevice);
	dim3 blocks = dim3(Count/BlockSize,1,1);
	dim3 threads = dim3(BlockSize,1,1);
	for (int i = 0; i < N; i++)
		Boris_kernel<<<blocks, threads>>> (partsDev, E, B, dt);

	hipDeviceSynchronize();

	hipMemcpy(parts, partsDev, nb, hipMemcpyDeviceToHost);

	hipFree(partsDev);
}