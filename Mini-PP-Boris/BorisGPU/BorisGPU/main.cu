#include "hip/hip_runtime.h"
#include "BorisGPU.cuh"
#include <iostream>
using namespace std;


int main()
{
	/*
	setlocale(LC_ALL, "Russian");
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("��� ����������: %s\n", deviceProp.name);
	printf("���� ���������� ������� � �����: %d\n", deviceProp.maxThreadsPerBlock);
	printf("������������ ����������� ������: x = %d, y = %d, z = %d\n",
      deviceProp.maxThreadsDim[0],
      deviceProp.maxThreadsDim[1],
      deviceProp.maxThreadsDim[2]);
	printf("������������ ������ �����: x = %d, y = %d, z = %d\n", 
      deviceProp.maxGridSize[0], 
      deviceProp.maxGridSize[1], 
      deviceProp.maxGridSize[2]); 
	 */


	const int Count = 1000000;
	double m = 9.10938291e-8;
	double q = -4.80320427e-5;
	Vector3 r = Vector3(1,0,0);
	Vector3 V = Vector3(0,2,0);
	Vector3 E = Vector3(1,2,3);
	Vector3 B = Vector3(1,2,3);
	
	Particle *W = new Particle[Count];
	for (int i=0; i<Count; i++)
		W[i] = Particle(r,V,m,q);

	double dt = 3;
	int N = 1000;
	int BlockSize = 1000;
	unsigned int start =  clock(); 
	BorisGPU(W,E,B,dt,N,Count,BlockSize);
	unsigned int end = clock(); 
	unsigned int time = end - start;
	
	cout << "Time: " << time/1000.0 << endl;

	/*for (int i=0; i<Count; i++)
	{
		cout << '(' << W[i].r.x << ", " << W[i].r.y << ", " << W[i].r.z << ')';
		cout << ' ';
		cout << '(' << W[i].V.x << ", " << W[i].V.y << ", " << W[i].V.z << ')' << endl;
	}*/

	cout << '(' << W[Count-1].r.x << ", " << W[Count-1].r.y << ", " << W[Count-1].r.z << ')';
	cout << ' ';
	cout << '(' << W[Count-1].V.x << ", " << W[Count-1].V.y << ", " << W[Count-1].V.z << ')' << endl;

	return 0;
}